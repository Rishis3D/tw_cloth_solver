#include "hip/hip_runtime.h"


//include
//-----------------------------------------------
//cuda
#include "hip/hip_vector_types.h"

//own
#include "cudaFunctionality.h"


//kernels
//-----------------------------------------------

namespace kernels
{
	//add_on_gpu
	__global__ void add_on_gpu(float a, float b, float* p_result)
	{
		//result
		*p_result = a + b;
	}
	
	//verletIntegration
	__global__ void verletIntegration(staticSolverData_t staticSolverData, dynamicSolverData_t dynamicSolverData)
	{
		//thread_index
		int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
		
		//execute if thread_index < vertexCount
		if(thread_index < staticSolverData.vertexCount)
		{
			
			//vecNew
			float3 vecNew = make_float3(0,0,0); //initialize to zero

			//vecPos
			float3 vecPos = make_float3(staticSolverData.pVertexPositionList[thread_index].x,
										staticSolverData.pVertexPositionList[thread_index].y,
										staticSolverData.pVertexPositionList[thread_index].z
										);

			//vecOldpos
			float3 vecOldpos = make_float3(staticSolverData.pVertexOldpositionList[thread_index].x,
										staticSolverData.pVertexOldpositionList[thread_index].y,
										staticSolverData.pVertexOldpositionList[thread_index].z
										);

			//vecForce
			float3 vecForce = make_float3(dynamicSolverData.pVertexForceList[thread_index].x,
										dynamicSolverData.pVertexForceList[thread_index].y,
										dynamicSolverData.pVertexForceList[thread_index].z
										);

			//drag
			float drag = 1.0 - dynamicSolverData.drag;

			//integration
			vecNew = (vecPos + ((vecPos - vecOldpos) * drag)) + vecForce;


			//update solverData

			//pVertexOldpositionList
			staticSolverData.pVertexOldpositionList[thread_index].x = vecPos.x;
			staticSolverData.pVertexOldpositionList[thread_index].y = vecPos.y;
			staticSolverData.pVertexOldpositionList[thread_index].z = vecPos.z;

			//pVertexOldpositionList
			staticSolverData.pVertexPositionList[thread_index].x = vecNew.x;
			staticSolverData.pVertexPositionList[thread_index].y = vecNew.y;
			staticSolverData.pVertexPositionList[thread_index].z = vecNew.z;

		}
	}

	//collisionConstraintGroundplane
	__global__ void collisionConstraintGroundplane(staticSolverData_t staticSolverData, dynamicSolverData_t dynamicSolverData)
	{
		//thread_index
		int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
		
		//execute if thread_index < vertexCount
		if(thread_index < staticSolverData.vertexCount)
		{
			//if point pos y is < than groundplane height set to groundplane height
			if(staticSolverData.pVertexPositionList[thread_index].y < dynamicSolverData.groundplaneHeight)
				staticSolverData.pVertexPositionList[thread_index].y = dynamicSolverData.groundplaneHeight;
		}
	}
	
	//collisionConstraintSpheres
	__global__ void collisionConstraintSpheres(staticSolverData_t staticSolverData, dynamicSolverData_t dynamicSolverData)
	{
		//thread_index
		int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
		
		//execute if thread_index < vertexCount
		if(thread_index < staticSolverData.vertexCount)
		{
			for(int collisionSphereIndex = 0; collisionSphereIndex < dynamicSolverData.collisionConstraintSpheresCount * 2; collisionSphereIndex+=2 )
			{
				
				//New
				//-----------------------------------------------

				//vecDownToUp and radiusSphere

				//vecUp
				float3 vecUp = make_float3(dynamicSolverData.pCollisionConstraintSpheresVecUpVecDownList[collisionSphereIndex].x,
											dynamicSolverData.pCollisionConstraintSpheresVecUpVecDownList[collisionSphereIndex].y,
											dynamicSolverData.pCollisionConstraintSpheresVecUpVecDownList[collisionSphereIndex].z
											);

				//vecDown
				float3 vecDown = make_float3(dynamicSolverData.pCollisionConstraintSpheresVecUpVecDownList[collisionSphereIndex+1].x,
											dynamicSolverData.pCollisionConstraintSpheresVecUpVecDownList[collisionSphereIndex+1].y,
											dynamicSolverData.pCollisionConstraintSpheresVecUpVecDownList[collisionSphereIndex+1].z
											);

				//vecDownToUp
				float3 vecDownToUp = make_float3(0,0,0);
				vecDownToUp = vecUp - vecDown;

				//vecSphereCenter
				float3 vecSphereCenter = make_float3(0,0,0);
				vecSphereCenter = vecDown + (vecDownToUp * 0.5);

				//radiusSphere
				float radiusSphere = length(vecDownToUp * 0.5);

				
				
				
				//perform collision projection

				//vecCurrentPoint
				float3 vecCurrentPoint = make_float3(staticSolverData.pVertexPositionList[thread_index].x,
													staticSolverData.pVertexPositionList[thread_index].y,
													staticSolverData.pVertexPositionList[thread_index].z
													);

				//vecSphereCenterToCurrentPoint
				float3 vecSphereCenterToCurrentPoint = vecCurrentPoint - vecSphereCenter;				
				
				
				//vecSphereCenterToCurrentPointLength
				float vecSphereCenterToCurrentPointLength = length(vecSphereCenterToCurrentPoint);
				
				//check if length < radius
				if(vecSphereCenterToCurrentPointLength < radiusSphere)
				{
					//currentPointProjected
					float3 currentPointProjected = make_float3(0,0,0);
					currentPointProjected = vecSphereCenter + ((vecSphereCenterToCurrentPoint / vecSphereCenterToCurrentPointLength) * radiusSphere);

					//set to solverData
					staticSolverData.pVertexPositionList[thread_index].x = currentPointProjected.x;
					staticSolverData.pVertexPositionList[thread_index].y = currentPointProjected.y;
					staticSolverData.pVertexPositionList[thread_index].z = currentPointProjected.z;

				};

			}
		}
	}
	
	//stickConstraint
	__global__ void stickConstraint(staticSolverData_t staticSolverData, dynamicSolverData_t dynamicSolverData)
	{
		//New

		//iterate edges
		for(int index = 0; index < staticSolverData.edgeCount; index++)
		{
			//restLength
			float restLength = staticSolverData.pEdgeRestlengthList[index];
			//vertexIndices
			int vertexIndex1 = int(staticSolverData.pEdgeVertexIndexList[index].x);
			int vertexIndex2 = int(staticSolverData.pEdgeVertexIndexList[index].y);
			
			//vec1
			float3 vec1 = make_float3(staticSolverData.pVertexPositionList[vertexIndex1].x,
										staticSolverData.pVertexPositionList[vertexIndex1].y,
										staticSolverData.pVertexPositionList[vertexIndex1].z
										);
			
			//vec2
			float3 vec2 = make_float3(staticSolverData.pVertexPositionList[vertexIndex2].x,
										staticSolverData.pVertexPositionList[vertexIndex2].y,
										staticSolverData.pVertexPositionList[vertexIndex2].z
										);

			//Compute new vector positions
			float3 delta = make_float3(0,0,0);
			delta = vec2 - vec1;
			float deltaLength = length(delta);
			float difference = (deltaLength - restLength) / deltaLength;
			vec1 = vec1 + delta * 0.5 * difference;
			vec2 = vec2 - delta * 0.5 * difference;

			//set vertexPositionList
			staticSolverData.pVertexPositionList[vertexIndex1].x = vec1.x;
			staticSolverData.pVertexPositionList[vertexIndex1].y = vec1.y;
			staticSolverData.pVertexPositionList[vertexIndex1].z = vec1.z;
			
			staticSolverData.pVertexPositionList[vertexIndex2].x = vec2.x;
			staticSolverData.pVertexPositionList[vertexIndex2].y = vec2.y;
			staticSolverData.pVertexPositionList[vertexIndex2].z = vec2.z;
		};
	};

	//positionConstraints
	__global__ void positionConstraints(staticSolverData_t staticSolverData, dynamicSolverData_t dynamicSolverData)
	{
		//check if posCons 
		if(dynamicSolverData.positionConstraintCount)
		{
			//iterate posCons
			for(int index = 0; index < dynamicSolverData.positionConstraintCount; index++)
			{
				//check if posCon active
				if(dynamicSolverData.pPositionConstraintActiveList[index])
				{
					//check if vertexIndex in vertexCount
					if(dynamicSolverData.pPositionConstraintVertexIndexList[index] < staticSolverData.vertexCount)
					{
						//set vertexPositionList
						staticSolverData.pVertexPositionList[dynamicSolverData.pPositionConstraintVertexIndexList[index]] = dynamicSolverData.pPositionConstraintCoordinateList[index];
					}
				}
			}
		}
	};
}



//definitions
//-----------------------------------------------

//cudaAvailable
int cuda::cudaAvailable()
{
	int deviceCount = 0;
	hipError_t cudaErrorId;
	cudaErrorId = hipGetDeviceCount(&deviceCount);
	return deviceCount;
};

//get_grid_size
int cuda::get_grid_size(const staticSolverData_t& staticSolverData, 
						const dynamicSolverData_t& dynamicSolverData, 
						grid_size_t grid_type = VERTEXCOUNT)
{
	//declare grid_count
	int grid_count;

	//thread_count
	int thread_count = get_thread_count(dynamicSolverData);
	
	//VERTEXCOUNT
	if(grid_type == VERTEXCOUNT)
	{
		//grid_count
		grid_count = (staticSolverData.vertexCount + (thread_count - 1)) / thread_count;
	}
	//EDGECOUNT
	else if(grid_type == EDGECOUNT)
	{
		//grid_count
		grid_count = (staticSolverData.edgeCount + (thread_count - 1)) / thread_count;
	};

	return grid_count;
	
};

//get_thread_count
int cuda::get_thread_count(const dynamicSolverData_t& dynamicSolverData)
{
	return dynamicSolverData.thread_count;
};

//add_on_gpu
float cuda::add_on_gpu(float a, float b)
{
	//size
	size_t size = sizeof(float);

	//d_p_result
	float* d_p_result;
	hipMalloc(&d_p_result, size);

	//h_p_result
	float* h_p_result = new float[1];
	
	//Call kernel
	dim3 num_blocks(1);
	dim3 threads_per_block(1);
	kernels::add_on_gpu<<<num_blocks, threads_per_block>>>(a, b, d_p_result);

	//copy
	hipMemcpy(h_p_result, d_p_result, size, hipMemcpyDeviceToHost);

	//free memory
	hipFree(d_p_result);
	
	//return
	return *h_p_result;

};


//set_static_solver_data_gpu
void cuda::set_static_solver_data_gpu(staticSolverData_t &staticSolverDataGPU, const staticSolverData_t &staticSolverData)
{
	
	//free mem if not null pointers
	//-----------------------------------------------
	//pVertexPositionList
	if(staticSolverDataGPU.pVertexPositionList)
		hipFree(staticSolverDataGPU.pVertexPositionList);
	//pVertexOldpositionList
	if(staticSolverDataGPU.pVertexOldpositionList)
		hipFree(staticSolverDataGPU.pVertexOldpositionList);
	//pVertexInitialpositionList
	if(staticSolverDataGPU.pVertexInitialpositionList)
		hipFree(staticSolverDataGPU.pVertexInitialpositionList);
	//pEdgeVertexIndexList
	if(staticSolverDataGPU.pEdgeVertexIndexList)
		hipFree(staticSolverDataGPU.pEdgeVertexIndexList);
	//pEdgeRestlengthList
	if(staticSolverDataGPU.pEdgeRestlengthList)
		hipFree(staticSolverDataGPU.pEdgeRestlengthList);

	
	
	
	//reallocate mem for pointers and copy data
	//-----------------------------------------------
	
	//size_vertex_vcs_vector
	size_t size_vertex_vcs_vector = staticSolverData.vertexCount * sizeof(vcsVector);
	//size_edge_vcs_vector
	size_t size_edge_vcs_vector = staticSolverData.edgeCount * sizeof(vcsVector);
	//size_float
	size_t size_float = staticSolverData.edgeCount * sizeof(float);

	//allocate
	hipMalloc(&staticSolverDataGPU.pVertexPositionList, size_vertex_vcs_vector);
	hipMalloc(&staticSolverDataGPU.pVertexOldpositionList, size_vertex_vcs_vector);
	hipMalloc(&staticSolverDataGPU.pVertexInitialpositionList, size_vertex_vcs_vector);
	hipMalloc(&staticSolverDataGPU.pEdgeVertexIndexList, size_edge_vcs_vector);
	hipMalloc(&staticSolverDataGPU.pEdgeRestlengthList, size_float);

	//copy
	//pVertexPositionList
	hipMemcpy(staticSolverDataGPU.pVertexPositionList, 
				staticSolverData.pVertexPositionList, 
				size_vertex_vcs_vector, 
				hipMemcpyHostToDevice);
	//pVertexOldpositionList
	hipMemcpy(staticSolverDataGPU.pVertexOldpositionList, 
				staticSolverData.pVertexOldpositionList, 
				size_vertex_vcs_vector, 
				hipMemcpyHostToDevice);
	//pVertexInitialpositionList
	hipMemcpy(staticSolverDataGPU.pVertexInitialpositionList, 
				staticSolverData.pVertexInitialpositionList, 
				size_vertex_vcs_vector, 
				hipMemcpyHostToDevice);
	//pEdgeVertexIndexList
	hipMemcpy(staticSolverDataGPU.pEdgeVertexIndexList, 
				staticSolverData.pEdgeVertexIndexList, 
				size_edge_vcs_vector, 
				hipMemcpyHostToDevice);
	//pEdgeRestlengthList
	hipMemcpy(staticSolverDataGPU.pEdgeRestlengthList, 
				staticSolverData.pEdgeRestlengthList, 
				size_float, 
				hipMemcpyHostToDevice);


	//set non pointer values
	//-----------------------------------------------
	staticSolverDataGPU.edgeCount = staticSolverData.edgeCount;
	staticSolverDataGPU.vertexCount = staticSolverData.vertexCount;

	
	//tmp
	//std::cout <<"Allocated static data on the device" <<std::endl;
};


//set_dynamic_solver_data_gpu
void cuda::set_dynamic_solver_data_gpu(dynamicSolverData_t &dynamicSolverDataGPU, 
										const dynamicSolverData_t &dynamicSolverData, 
										const staticSolverData_t &staticSolverData)
{
	
	
	//free mem if not null pointers
	//-----------------------------------------------
	//pVertexForceList
	if(dynamicSolverDataGPU.pVertexForceList)
		hipFree(dynamicSolverDataGPU.pVertexForceList);
	//pPositionConstraintActiveList
	if(dynamicSolverDataGPU.pPositionConstraintActiveList)
		hipFree(dynamicSolverDataGPU.pPositionConstraintActiveList);
	//pPositionConstraintVertexIndexList
	if(dynamicSolverDataGPU.pPositionConstraintVertexIndexList)
		hipFree(dynamicSolverDataGPU.pPositionConstraintVertexIndexList);
	//pPositionConstraintCoordinateList
	if(dynamicSolverDataGPU.pPositionConstraintCoordinateList)
		hipFree(dynamicSolverDataGPU.pPositionConstraintCoordinateList);
	//pCollisionConstraintSpheresVecUpVecDownList
	if(dynamicSolverDataGPU.pCollisionConstraintSpheresVecUpVecDownList)
		hipFree(dynamicSolverDataGPU.pCollisionConstraintSpheresVecUpVecDownList);
	//pCollisionConstraintConvexTriangleCountList
	if(dynamicSolverDataGPU.pCollisionConstraintConvexTriangleCountList)
		hipFree(dynamicSolverDataGPU.pCollisionConstraintConvexTriangleCountList);
	//pCollisionConstraintConvexTriangleVertexPositionList
	if(dynamicSolverDataGPU.pCollisionConstraintConvexTriangleVertexPositionList)
		hipFree(dynamicSolverDataGPU.pCollisionConstraintConvexTriangleVertexPositionList);
	

	
	
	
	//reallocate mem for pointers and copy data
	//-----------------------------------------------
	
	//size_vertex_vcs_vector
	size_t size_vertex_vcs_vector = staticSolverData.vertexCount * sizeof(vcsVector);
	//size_position_constraint_int
	size_t size_position_constraint_int = dynamicSolverData.positionConstraintCount * sizeof(int);
	//size_position_constraint_vcs_vector
	size_t size_position_constraint_vcs_vector = dynamicSolverData.positionConstraintCount * sizeof(vcsVector);
	//size_collision_constraint_spheres_vcs_vector
	size_t size_collision_constraint_spheres_vcs_vector = dynamicSolverData.collisionConstraintSpheresCount * 2 * sizeof(vcsVector);
	//size_collision_constraint_convex_int
	size_t size_collision_constraint_convex_int = dynamicSolverData.collisionConstraintConvexCount * sizeof(int);
	//size_collision_constraint_convex_vcs_vector
	size_t size_collision_constraint_convex_vcs_vector = dynamicSolverData.collisionConstraintConvexTriangleVertexCount * sizeof(vcsVector);
	

	//allocate
	hipMalloc(&dynamicSolverDataGPU.pVertexForceList, size_vertex_vcs_vector);
	
	hipMalloc(&dynamicSolverDataGPU.pPositionConstraintActiveList, size_position_constraint_int);
	hipMalloc(&dynamicSolverDataGPU.pPositionConstraintVertexIndexList, size_position_constraint_int);
	hipMalloc(&dynamicSolverDataGPU.pPositionConstraintCoordinateList, size_position_constraint_vcs_vector);

	hipMalloc(&dynamicSolverDataGPU.pCollisionConstraintSpheresVecUpVecDownList, size_collision_constraint_spheres_vcs_vector);

	hipMalloc(&dynamicSolverDataGPU.pCollisionConstraintConvexTriangleCountList, size_collision_constraint_convex_int);
	hipMalloc(&dynamicSolverDataGPU.pCollisionConstraintConvexTriangleVertexPositionList, size_collision_constraint_convex_vcs_vector);

	

	//copy
	//pVertexForceList
	hipMemcpy(dynamicSolverDataGPU.pVertexForceList, 
				dynamicSolverData.pVertexForceList, 
				size_vertex_vcs_vector, 
				hipMemcpyHostToDevice);
	
	//pPositionConstraintActiveList
	hipMemcpy(dynamicSolverDataGPU.pPositionConstraintActiveList, 
				dynamicSolverData.pPositionConstraintActiveList, 
				size_position_constraint_int, 
				hipMemcpyHostToDevice);
	//pPositionConstraintVertexIndexList
	hipMemcpy(dynamicSolverDataGPU.pPositionConstraintVertexIndexList, 
				dynamicSolverData.pPositionConstraintVertexIndexList, 
				size_position_constraint_int, 
				hipMemcpyHostToDevice);
	//pPositionConstraintCoordinateList
	hipMemcpy(dynamicSolverDataGPU.pPositionConstraintCoordinateList, 
				dynamicSolverData.pPositionConstraintCoordinateList, 
				size_position_constraint_vcs_vector, 
				hipMemcpyHostToDevice);

	//pCollisionConstraintSpheresVecUpVecDownList
	hipMemcpy(dynamicSolverDataGPU.pCollisionConstraintSpheresVecUpVecDownList, 
				dynamicSolverData.pCollisionConstraintSpheresVecUpVecDownList, 
				size_collision_constraint_spheres_vcs_vector, 
				hipMemcpyHostToDevice);

	//pCollisionConstraintConvexTriangleCountList
	hipMemcpy(dynamicSolverDataGPU.pCollisionConstraintConvexTriangleCountList, 
				dynamicSolverData.pCollisionConstraintConvexTriangleCountList, 
				size_collision_constraint_convex_int, 
				hipMemcpyHostToDevice);

	//pCollisionConstraintConvexTriangleVertexPositionList
	hipMemcpy(dynamicSolverDataGPU.pCollisionConstraintConvexTriangleVertexPositionList, 
				dynamicSolverData.pCollisionConstraintConvexTriangleVertexPositionList, 
				size_collision_constraint_convex_vcs_vector, 
				hipMemcpyHostToDevice);
	

	
	//set non pointer values
	//-----------------------------------------------
	dynamicSolverDataGPU.drag = dynamicSolverData.drag;
	dynamicSolverDataGPU.collisionGroundplaneActive = dynamicSolverData.collisionGroundplaneActive;
	dynamicSolverDataGPU.groundplaneHeight = dynamicSolverData.groundplaneHeight;
	dynamicSolverDataGPU.repetitions = dynamicSolverData.repetitions;
	dynamicSolverDataGPU.positionConstraintCount = dynamicSolverData.positionConstraintCount;
	dynamicSolverDataGPU.collisionConstraintSpheresCount = dynamicSolverData.collisionConstraintSpheresCount;
	dynamicSolverDataGPU.collisionConstraintConvexCount = dynamicSolverData.collisionConstraintConvexCount;
	dynamicSolverDataGPU.collisionConstraintConvexTriangleVertexCount = dynamicSolverData.collisionConstraintConvexTriangleVertexCount;
	dynamicSolverDataGPU.collisionConstraintConvexOffset = dynamicSolverData.collisionConstraintConvexOffset;
	
	dynamicSolverDataGPU.thread_count = dynamicSolverData.thread_count;
	dynamicSolverDataGPU.omp = dynamicSolverData.omp;


	//tmp
	//std::cout <<"Allocated dynamic data on the device" <<std::endl;
};

//set_static_solver_data_from_gpu
void cuda::set_static_solver_data_from_gpu(staticSolverData_t& staticSolverData, staticSolverData_t& staticSolverDataGPU)
{
	//size_vertex_vcs_vector
	size_t size_vertex_vcs_vector = staticSolverData.vertexCount * sizeof(vcsVector);
	//size_edge_vcs_vector
	size_t size_edge_vcs_vector = staticSolverData.edgeCount * sizeof(vcsVector);
	//size_float
	size_t size_float = staticSolverData.edgeCount * sizeof(float);
	
	
	//copy
	//pVertexPositionList
	hipMemcpy(staticSolverData.pVertexPositionList, 
				staticSolverDataGPU.pVertexPositionList, 
				size_vertex_vcs_vector, 
				hipMemcpyDeviceToHost);
	//pVertexOldpositionList
	hipMemcpy(staticSolverData.pVertexOldpositionList, 
				staticSolverDataGPU.pVertexOldpositionList, 
				size_vertex_vcs_vector, 
				hipMemcpyDeviceToHost);
	//pVertexInitialpositionList
	hipMemcpy(staticSolverData.pVertexInitialpositionList, 
				staticSolverDataGPU.pVertexInitialpositionList, 
				size_vertex_vcs_vector, 
				hipMemcpyDeviceToHost);
	//pEdgeVertexIndexList
	hipMemcpy(staticSolverData.pEdgeVertexIndexList, 
				staticSolverDataGPU.pEdgeVertexIndexList, 
				size_edge_vcs_vector, 
				hipMemcpyDeviceToHost);
	//pEdgeRestlengthList
	hipMemcpy(staticSolverData.pEdgeRestlengthList, 
				staticSolverDataGPU.pEdgeRestlengthList, 
				size_float, 
				hipMemcpyDeviceToHost);
};

//verletIntegration
void cuda::verletIntegration(staticSolverData_t &staticSolverData, dynamicSolverData_t &dynamicSolverData)
{
	
	//thread_count
	dim3 thread_count(get_thread_count(dynamicSolverData));
	//grid_size_vertex
	dim3 grid_size_vertex(get_grid_size(staticSolverData, dynamicSolverData));
	
	//verletIntegration
	kernels::verletIntegration<<<grid_size_vertex, thread_count>>>(staticSolverData, dynamicSolverData);
	
	//log
	//std::cout<<"Cuda::verletIntegrationGPU"<<std::endl;

	
};

//collisionConstraintGroundplane
void cuda::collisionConstraintGroundplane(staticSolverData_t &staticSolverData, dynamicSolverData_t &dynamicSolverData)
{
	//thread_count
	dim3 thread_count(get_thread_count(dynamicSolverData));
	//grid_size_vertex
	dim3 grid_size_vertex(get_grid_size(staticSolverData, dynamicSolverData));

	//kernel
	kernels::collisionConstraintGroundplane<<<grid_size_vertex, thread_count>>> (staticSolverData, dynamicSolverData);
	
	//log
	//std::cout<<"Cuda::collisionConstraintGroundplane"<<std::endl;

	
};

//collisionConstraintSpheres
void cuda::collisionConstraintSpheres(staticSolverData_t &staticSolverData, dynamicSolverData_t &dynamicSolverData)
{
	//thread_count
	dim3 thread_count(get_thread_count(dynamicSolverData));
	//grid_size_vertex
	dim3 grid_size_vertex(get_grid_size(staticSolverData, dynamicSolverData));

	//kernel
	kernels::collisionConstraintSpheres<<<grid_size_vertex, thread_count>>> (staticSolverData, dynamicSolverData);
	
	//log
	//std::cout<<"Cuda::collisionConstraintGroundplane"<<std::endl;

	
};

//collisionConstraintConvex
void cuda::collisionConstraintConvex(staticSolverData_t &staticSolverData, dynamicSolverData_t &dynamicSolverData)
{
	//log
	std::cout<<"Cuda::collisionConstraintConvex Dummy"<<std::endl;
};

//stickConstraint
void cuda::stickConstraint(staticSolverData_t &staticSolverData, dynamicSolverData_t &dynamicSolverData)
{
	//kernel (serial since dependent on serial execution)
	kernels::stickConstraint<<<1,1>>> (staticSolverData, dynamicSolverData);
};

//positionConstraints
void cuda::positionConstraints(staticSolverData_t &staticSolverData, dynamicSolverData_t &dynamicSolverData)
{
	//kernel (serial since dependent on serial execution)
	kernels::positionConstraints<<<1,1>>> (staticSolverData, dynamicSolverData);
};